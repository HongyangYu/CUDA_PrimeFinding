#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

typedef unsigned long long int uint64_t;
/******************************************************************************
* kernel for finding prime numbers using the sieve of eratosthenes
* - primes: an array of bools. initially all numbers are set to "0".
*			  A "0" value means that the number at that index is prime.
* - max: the max size of the primes array
******************************************************************************/
__global__ static void sieveOfEratosthenesCUDA(char *primes, uint64_t max) {
	// first thread 0
	if (threadIdx.x == 0 && threadIdx.y == 0){
		primes[0] = 1; // value of 1 means the number is NOT prime
		primes[1] = 1; // numbers "0" and "1" are not prime numbers
		// sieve multiples of two
		for (int j = 2 * 2; j < max; j += 2){
			primes[j] = 1;
		}
	} else {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		const uint64_t maxRoot = sqrt((double)max);
		// make sure index won't go out of bounds, 
		// also don't execute it on index 1
		if (index < maxRoot && primes[index] == 0 && index > 1 ){
			// mark off the composite numbers
			for (int j = index * index; j < max; j += index){
				primes[j] = 1;
			}
		}
	}
}
/*******************************************************************************
 * checkDevice()
 ******************************************************************************/
__host__ int checkDevice(){
	printf("Checking device...\n");
	// query the Device and decide on the block size
	int devID = 0; // the default device ID
	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);
	if (error != hipSuccess){
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	error = hipGetDeviceProperties(&deviceProp, devID);
	if (deviceProp.computeMode == hipComputeModeProhibited || error != hipSuccess){
		printf("CUDA device ComputeMode is prohibited or failed to getDeviceProperties\n");
		return EXIT_FAILURE;
	}
	// Use a larger block size for Fermi and above (see compute capability)
	return (deviceProp.major < 2) ? 16 : 32;
}
/******************************************************************************
* genPrimesOnDevice
* - inputs: limit - the largest prime that should be computed
*			primes - an array of size [limit], initialized to 0
******************************************************************************/
__host__ void genPrimesOnDevice(char* primes, uint64_t max){
	int blockSize = checkDevice();
	if (blockSize == EXIT_FAILURE)
		return;
	char* d_Primes = NULL;
	int sizePrimes = sizeof(char) * max;
	uint64_t maxRoot = sqrt(max);
	// allocate the primes on the device and set them to 0
	hipMalloc(&d_Primes, sizePrimes);
	hipMemset(d_Primes, 0, sizePrimes);
	// make sure that there are no errors...
	hipPeekAtLastError();
	// setup the execution configuration
	dim3 dimBlock(maxRoot, 1, 1);
	dim3 dimGrid(1);
	//////// debug
	#ifdef DEBUG
	printf("dimBlock(%d, %d, %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
	printf("dimGrid(%d, %d, %d)\n", dimGrid.x, dimGrid.y, dimGrid.z);
	#endif
	// call the kernel
	sieveOfEratosthenesCUDA<<<dimGrid, dimBlock>>>(d_Primes, max);
	// check for kernel errors
	hipPeekAtLastError();
	hipDeviceSynchronize();
	// copy the results back
	hipMemcpy(primes, d_Primes, sizePrimes, hipMemcpyDeviceToHost);
	// no memory leaks
	hipFree(d_Primes);
}
/**********************************************************************************/
int main(){
	uint64_t maxPrime = 102; // find all primes from 0 to N-1
	char* primes = (char*) malloc(maxPrime);
	memset(primes, 0, maxPrime); // initialize all elements to 0
	genPrimesOnDevice(primes, maxPrime);
	// display the results
	int i;
	for (i = 0; i < maxPrime; i++)
		if (primes[i] == 0)
			printf("%i ", i);
	printf("\n");
	free(primes);
	return 0;
}
